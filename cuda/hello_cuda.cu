#include <stdio.h>
#include <hip/hip_runtime.h>

/* Device code: runs on GPU */
__global__ void Hello(void) {

    printf("Hello from thread %d!\n", threadIdx.x);

}


/* Host code: runs on CPU */
int main(int argc, char *argv[]) {
    int thread_count;
    thread_count = strtol(argv[1],NULL, 10);

    Hello <<<2, thread_count/2 >>>();

    hipDeviceSynchronize(); //wait for GPU to finish
    
    return 0; 

}